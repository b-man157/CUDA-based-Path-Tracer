#include "hip/hip_runtime.h"
#include "lambertian.hpp"
#include "rtweekend.hpp"

#include "camera.hpp"
#include "color.hpp"
#include "hittable_list.hpp"
#include "material.hpp"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <fstream>
#include <iostream>
#include <numeric>

__constant__ unsigned seed = 42;

__global__ void setup_curand(hiprandState *state, int image_width, int image_height) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < image_width && idy < image_height) {
        auto id = idy * image_width + idx;
        hiprand_init(seed, id, 0, &state[id]);
    }
}

template <int depth>
__device__ color ray_color(
        hiprandState *local_state, const ray &r, const hittable_list *world) {
    hit_record rec;
    // Fix shadow acne by setting t_min > 0.
    if (world->hit(r, 0.001, infinity, rec)) {
        ray scattered;
        color attenuation;
        if (rec.mat_ptr->scatter(local_state, r, rec, attenuation, scattered))
            return attenuation * ray_color<depth - 1>(local_state, scattered, world);
        return color(0, 0, 0);
    }

    vec3 unit_direction = unit_vector(r.direction());
    auto t = 0.5 * (unit_direction.y() + 1.0);
    return (1.0 - t)*color(1.0, 1.0, 1.0) + t*color(0.5, 0.7, 1.0);
}

template <>
__device__ color ray_color<0>(
        hiprandState *local_state, const ray &r, const hittable_list *world) {
    return color(0, 0, 0);
}

template <int max_depth>
__global__ void render(
        hiprandState *state, const camera *cam, int image_width, int image_height,
        const hittable_list *world, color *pixel_colors, int samples_per_pixel) {
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < image_width && idy < image_height) {
        auto id = idy * image_width + idx;
        auto p_index = (image_height-1 - idy) * image_width + idx;
        pixel_colors[p_index] = color(0, 0, 0);

        for (int i = 0; i < samples_per_pixel; ++i) {
            float u = (idx + random_float(&state[id])) / (image_width-1);
            float v = (idy + random_float(&state[id])) / (image_height-1);
            auto r = cam->get_ray(u, v);

            pixel_colors[p_index] += ray_color<max_depth>(&state[id], r, world);
        }
    }
}

int main(int argc, char **argv) {
    // Arguments

    if (argc != 2) {
        std::cerr << "Specify output file.\n";
        return -1;
    }

    // Image

    const float aspect_ratio = 16.0 / 9.0;
    const int image_width = 400;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    const int samples_per_pixel = 100;
    const int max_depth = 50;

    // Dimensions

    dim3 block_dim;
    block_dim.x = sqrt(1024 * aspect_ratio);
    block_dim.y = block_dim.x / aspect_ratio;
    block_dim.z = 1;

    dim3 grid_dim;
    grid_dim.x = (image_width  / block_dim.x) + (image_width  % block_dim.x > 0);
    grid_dim.y = (image_height / block_dim.y) + (image_height % block_dim.y > 0);
    grid_dim.z = 1;

    // World

    const size_t n_spheres = 5;
    point3 centers[n_spheres] = {
        {0, -100.5, -1},
        {0, 0, -1},
        {-1, 0, -1},
        {-1, 0, -1},
        {1, 0, -1}
    };
    float radii[n_spheres] = {100, 0.5, 0.5, -0.4, 0.5};
    material materials[n_spheres] = {
        lambertian(color(0.8, 0.8, 0.0)),
        lambertian(color(0.1, 0.2, 0.5)),
        dielectric(1.5),
        dielectric(1.5),
        metal(color(0.8, 0.6, 0.2), 0.0)
    };

    hittable_list h_world;
    h_world.add_spheres(n_spheres, centers, radii, materials);

    // Camera

    camera h_cam(point3(-2, 2, 1), point3(0, 0, -1), vec3(0, 1, 0), 20.0, aspect_ratio);

    camera *d_cam;
    hipMalloc(&d_cam, sizeof(camera));
    hipMemcpyAsync(d_cam, &h_cam, sizeof(camera), hipMemcpyHostToDevice);

    // Render

    std::ofstream f_out(argv[1]);

    f_out << "P3\n" << image_width << ' ' << image_height << "\n255";

    const int n_pixels = image_width * image_height;
    color *d_pixels;
    hipMalloc(&d_pixels, n_pixels * sizeof(color));

    hittable_list *d_world;
    hipMalloc(&d_world, sizeof(hittable_list));
    hipMemcpyAsync(d_world, &h_world, sizeof(hittable_list), hipMemcpyHostToDevice);

    hiprandState *d_state;
    hipMalloc(&d_state, n_pixels * sizeof(hiprandState));
    setup_curand<<<grid_dim, block_dim>>>(d_state, image_width, image_height);

    // TODO: Track and print progress.
    render<max_depth><<<grid_dim, block_dim>>>(
        d_state, d_cam, image_width, image_height, d_world, d_pixels, samples_per_pixel);

    h_world.clear();
    hipFree(d_state);
    hipFree(d_cam);
    hipFree(d_world);

    color *h_pixels = (color *) std::malloc(n_pixels * sizeof(color));
    hipMemcpy(h_pixels, d_pixels, n_pixels * sizeof(color), hipMemcpyDeviceToHost);
    hipFree(d_pixels);

    f_out << std::accumulate(h_pixels, h_pixels + n_pixels, std::string(""),
        [samples_per_pixel](const std::string s, const color c) {
            return s + '\n' + to_string(c, samples_per_pixel);
        }
    );
    std::free(h_pixels);

    f_out.close();

    return 0;
}
