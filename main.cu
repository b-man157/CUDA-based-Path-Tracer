#include "hip/hip_runtime.h"
#include "color.hpp"
#include "ray.hpp"
#include "vec3.hpp"

#include <hip/hip_runtime.h>

#include <cstdlib>
#include <fstream>
#include <iostream>
#include <numeric>

// TODO: Remove later when redundant.
#include <cmath>
using std::sqrt;

__device__ color ray_color(ray r) {
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5 * (unit_direction.y() + 1.0);

    return (1.0 - t)*color(1.0, 1.0, 1.0) + t*color(0.5, 0.7, 1.0);
}

// TODO: Better name?
__global__ void render(int i_width, int i_height, float v_width, float v_height, float f_length, color *pixel_colors) {
    __shared__ point3 origin, horizontal, vertical, lower_left_corner;

    if (!threadIdx.x && !threadIdx.y) {
        origin = point3(0, 0, 0);
        horizontal = vec3(v_width, 0, 0);
        vertical = vec3(0, v_height, 0);
        lower_left_corner = origin - horizontal/2 - vertical/2 - vec3(0, 0, f_length);
    }
    __syncthreads();

    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < i_width && idy < i_height) {
        float u = float(idx) / (i_width-1);
        float v = float(idy) / (i_height-1);
        ray r(origin, lower_left_corner + u*horizontal + v*vertical - origin);

        auto p_index = (i_height-1 - idy) * i_width + idx;
        pixel_colors[p_index] = ray_color(r);
    }
}

int main(int argc, char **argv) {
    // Arguments

    if (argc != 2) {
        std::cerr << "Specify output file.\n";
        return -1;
    }

    // Image

    const float aspect_ratio = 16.0 / 9.0;
    const int image_width  = 400;
    const int image_height = static_cast<int>(image_width / aspect_ratio);

    // Dimensions

    dim3 block_dim;
    block_dim.x = sqrt(1024 * aspect_ratio);
    block_dim.y = block_dim.x / aspect_ratio;
    block_dim.z = 1;

    dim3 grid_dim;
    grid_dim.x = (image_width  / block_dim.x) + (image_width  % block_dim.x > 0);
    grid_dim.y = (image_height / block_dim.y) + (image_height % block_dim.y > 0);
    grid_dim.z = 1;

    // Camera

    auto viewport_height = 2.0;
    auto viewport_width = aspect_ratio * viewport_height;
    auto focal_length = 1.0;

    // Render

    std::ofstream f_out(argv[1]);

    f_out << "P3\n" << image_width << ' ' << image_height << "\n255";

    const int n_pixels = image_width * image_height;
    color *d_pixel_colors, *h_pixel_colors = (color *) std::malloc(n_pixels * sizeof(color));
    hipMalloc(&d_pixel_colors, n_pixels * sizeof(color));

    // TODO: Track and print progress.
    render<<<grid_dim, block_dim>>>(image_width, image_height, viewport_width, viewport_height, focal_length, d_pixel_colors);
    hipMemcpy(h_pixel_colors, d_pixel_colors, n_pixels * sizeof(color), hipMemcpyDeviceToHost);

    f_out << std::accumulate(h_pixel_colors, h_pixel_colors + n_pixels, std::string(""),
        [](const std::string s, const color c) {
            return s + '\n' + to_string(c);
        }
    );

    f_out.close();

    return 0;
}
